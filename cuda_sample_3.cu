#include "hip/hip_runtime.h"
/*
 * cuda_sample_3.cu �������л�����Ԫ��ƽ���ͼ���
 * ÿ���̸߳����ۼ������е�һ����������Ԫ��
 *
 * @author chenyang li
 *
 */
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
// ���� threadIdx
#include <>
#include <time.h>

#define DATA_SIZE 1024*1024

// �߳���
#define THREAD_NUM 256

int data[DATA_SIZE];
int clockRate;

/* ����0-9֮�������� */
void generateNumbers(int *numbers, int size) {
    int i;
    for (i = 0; i < size; i++) {
        numbers[i] = rand() % 10;
    }
}

/* ��ӡGPU�豸��Ϣ */
void printDeviceProps(const hipDeviceProp_t *prop) {
    printf("Device Name: %s\n", prop->name);
    printf("totalGlobalMem: %ld\n", prop->totalGlobalMem);
    printf("sharedMemPerBlock: %d\n", prop->sharedMemPerBlock);
    printf("regsPerBlock: %d\n", prop->regsPerBlock);
    printf("warpSize: %d\n", prop->warpSize);
    printf("memPitch: %d\n", prop->memPitch);
    printf("maxThreadPerBlock: %d\n", prop->maxThreadsPerBlock);
    printf("maxThreadsDim[0-2]: %d %d %d\n", prop->maxThreadsDim[0], prop->maxThreadsDim[1], prop->maxThreadsDim[2]);
    printf("maxGridSize[0-2]: %d %d %d\n", prop->maxGridSize[0], prop->maxGridSize[1], prop->maxGridSize[2]);
    printf("totalConstMem: %d\n", prop->totalConstMem);
    printf("major: %d & minor: %d\n", prop->major, prop->minor);
    printf("clockRate: %d\n", prop->clockRate); clockRate = prop->clockRate;
    printf("textureAlignment: %d\n", prop->textureAlignment);
    printf("deviceOverlap: %d\n", prop->deviceOverlap);
    printf("multiProcessorCount: %d\n", prop->multiProcessorCount);
}

/* CUDA ��ʼ�� */
bool initCUDA() {
    int count, i;
    hipDeviceProp_t prop;

    hipGetDeviceCount(&count);

    if (0 == count) {
        fprintf(stderr, "There is no device.\n");
        return false;
    }

    for (i = 0; i < count; i++) {
        if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
            if (prop.major >= 1) {
                break;
            }
        }
    }

    if (i == count) {
        fprintf(stderr, "There is no device.\n");
        return false;
    }

    hipSetDevice(i);

    printDeviceProps(&prop);

    return true;
}

/* ����ƽ���ͣ�__global__����������GPU��*/
__global__ static void sumOfSquares(int *numbers, int *sub_sum, clock_t *time) {
    int i;
    clock_t start, end;

    // ��ȡ��ǰ�߳�Id����0��ʼ��
    const int thread_id = threadIdx.x;
    // ÿ���߳��ۼ�Ԫ�صĸ���
    const int size = DATA_SIZE / THREAD_NUM;

    // ��¼�߳�0����ʼʱ��
    if (thread_id == 0) {
        start = clock();
    }

    sub_sum[thread_id] = 0;
    for (i = thread_id * size; i < (thread_id + 1) * size; i++) {
        sub_sum[thread_id] += numbers[i] * numbers[i];
    }

    // ��¼�߳�0�Ľ���ʱ��
    if (thread_id == 0) {
        end = clock();
        *time = end - start;
    }
}

int main(void) {
    if (!initCUDA()) {
        return 0;
    }

    int *gpudata;
    int i, sum;
    int sub_sum[THREAD_NUM], *gpu_sub_sum;
    clock_t time_used, *gpu_time_used;

    generateNumbers(data, DATA_SIZE);

    hipMalloc((void**)&gpudata, sizeof(int) * DATA_SIZE);
    // ʹ�ó���ΪTHREAD_NUM����������¼ÿ���̼߳���Ľ��
    hipMalloc((void**)&gpu_sub_sum, sizeof(int) * THREAD_NUM);
    hipMalloc((void**)&gpu_time_used, sizeof(clock_t));

    hipMemcpy(gpudata, data, sizeof(int) * DATA_SIZE, hipMemcpyHostToDevice);

    // �����߳�����
    sumOfSquares << < 1, THREAD_NUM, 0 >> > (gpudata, gpu_sub_sum, gpu_time_used);

    hipMemcpy(&time_used, gpu_time_used, sizeof(clock_t), hipMemcpyDeviceToHost);
    // ���Դ��е����鿽�������ڴ���
    hipMemcpy(sub_sum, gpu_sub_sum, sizeof(int) * THREAD_NUM, hipMemcpyDeviceToHost);

    sum = 0;
    for (i = 0; i < THREAD_NUM; i++) {
        sum += sub_sum[i];
    }

    hipFree(gpudata);
    // �ͷ��Դ��е�����
    hipFree(gpu_sub_sum);
    hipFree(time);

    printf("\nGPU sum is: %d, time used: %f (s)\n", sum, (float)time_used / (clockRate * 1000));

    sum = 0;
    for (i = 0; i < DATA_SIZE; i++) {
        sum += data[i] * data[i];
    }
    printf("CPU sum is: %d\n", sum);
    printf("Memory bandwidth: %f (MB/s)\n", ((float)(DATA_SIZE * sizeof(int) / 1024 / 1024)) / ((float)time_used / (clockRate * 1000)));

    system("pause");

    // return 0;
}