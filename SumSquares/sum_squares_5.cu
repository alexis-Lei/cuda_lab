#include "hip/hip_runtime.h"
/*
 * sum_squares_5.cu �ڸĽ���ȡģʽ�Ļ����ϣ�����Block�����߳���������һ���Ż�����Ԫ��ƽ���ͼ���
 *
 * @author chenyang li
 */
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <>
#include <time.h>

#define DATA_SIZE 1024 * 1024
#define THREAD_NUM 216
// Block����
#define BLOCK_NUM 32

int data[DATA_SIZE];
int clockRate;

/* ����0-9֮�������� */
void generateNumbers(int *numbers, int size) {
    int i;
    for (i = 0; i < size; i++) {
        numbers[i] = rand() % 10;
    }
}

/* ��ӡGPU�豸��Ϣ */
void printDeviceProps(const hipDeviceProp_t *prop) {
    printf("Device Name: %s\n", prop->name);
    printf("totalGlobalMem: %ld\n", prop->totalGlobalMem);
    printf("sharedMemPerBlock: %d\n", prop->sharedMemPerBlock);
    printf("regsPerBlock: %d\n", prop->regsPerBlock);
    printf("warpSize: %d\n", prop->warpSize);
    printf("memPitch: %d\n", prop->memPitch);
    printf("maxThreadPerBlock: %d\n", prop->maxThreadsPerBlock);
    printf("maxThreadsDim[0-2]: %d %d %d\n", prop->maxThreadsDim[0], prop->maxThreadsDim[1], prop->maxThreadsDim[2]);
    printf("maxGridSize[0-2]: %d %d %d\n", prop->maxGridSize[0], prop->maxGridSize[1], prop->maxGridSize[2]);
    printf("totalConstMem: %d\n", prop->totalConstMem);
    printf("major: %d & minor: %d\n", prop->major, prop->minor);
    printf("clockRate: %d\n", prop->clockRate); clockRate = prop->clockRate;
    printf("textureAlignment: %d\n", prop->textureAlignment);
    printf("deviceOverlap: %d\n", prop->deviceOverlap);
    printf("multiProcessorCount: %d\n", prop->multiProcessorCount);
}

/* CUDA ��ʼ�� */
bool initCUDA() {
    int count, i;
    hipDeviceProp_t prop;

    hipGetDeviceCount(&count);

    if (0 == count) {
        fprintf(stderr, "There is no device.\n");
        return false;
    }

    for (i = 0; i < count; i++) {
        if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
            if (prop.major >= 1) {
                break;
            }
        }
    }

    if (i == count) {
        fprintf(stderr, "There is no device.\n");
        return false;
    }

    hipSetDevice(i);

    printDeviceProps(&prop);

    return true;
}

/* Ѱ�Һ�ʱ����Ԫ�� */
clock_t findMaxTime(clock_t *time, int size) {
    int i;
    clock_t max = time[0];
    for (i = 0; i < size; i++) {
        if (time[i] > max) {
            max = time[i];
        }
    }
    return max;
}

/* ����ƽ���ͣ�__global__����������GPU��*/
__global__ static void sumOfSquares(int *numbers, int *sub_sum, clock_t *time) {
    int i;
    clock_t start, end;

    // ��ȡ��ǰ�߳�������Block�ţ���0��ʼ��
    const int block_id = blockIdx.x;
    const int thread_id = threadIdx.x;

    if (thread_id == 0) {
        start = clock();
    }

    sub_sum[block_id * THREAD_NUM + thread_id] = 0;
    // Block0-�߳�0��ȡ��0��Ԫ�أ�Block0-�߳�1��ȡ��1��Ԫ��...Block1-�߳�0��ȡ��THREAD_NUM��Ԫ�أ��Դ�����... 
    for (i = block_id * THREAD_NUM + thread_id; i < DATA_SIZE; i += BLOCK_NUM * THREAD_NUM) {
        sub_sum[block_id * THREAD_NUM + thread_id] += numbers[i] * numbers[i];
    }

    if (thread_id == 0) {
        end = clock();
        time[block_id] = end - start;
    }
}

int main(void) {
    if (!initCUDA()) {
        return 0;
    }

    int *gpudata;
    int i, sum;
    int sub_sum[BLOCK_NUM * THREAD_NUM], *gpu_sub_sum;
    // ÿ��Block����һ����ʱ��Ԫ
    clock_t time_used[BLOCK_NUM], *gpu_time_used;

    generateNumbers(data, DATA_SIZE);

    hipMalloc((void**)&gpudata, sizeof(int) * DATA_SIZE);
    // ��ǰһ����BLOCK_NUM * THREAD_NUM���߳�
    hipMalloc((void**)&gpu_sub_sum, sizeof(int) * BLOCK_NUM * THREAD_NUM);
    hipMalloc((void**)&gpu_time_used, sizeof(clock_t) * BLOCK_NUM);

    hipMemcpy(gpudata, data, sizeof(int) * DATA_SIZE, hipMemcpyHostToDevice);

    // ����Block��
    sumOfSquares << < BLOCK_NUM, THREAD_NUM, 0 >> > (gpudata, gpu_sub_sum, gpu_time_used);

    hipMemcpy(time_used, gpu_time_used, sizeof(clock_t) * BLOCK_NUM, hipMemcpyDeviceToHost);
    hipMemcpy(sub_sum, gpu_sub_sum, sizeof(int) * BLOCK_NUM * THREAD_NUM, hipMemcpyDeviceToHost);

    sum = 0;
    for (i = 0; i < BLOCK_NUM * THREAD_NUM; i++) {
        sum += sub_sum[i];
    }

    hipFree(gpudata);
    hipFree(gpu_sub_sum);
    hipFree(time);

    clock_t max_time_used = findMaxTime(time_used, BLOCK_NUM);
    printf("\nGPU sum is: %d, time used: %f (s)\n", sum, (float)max_time_used / (clockRate * 1000));

    sum = 0;
    for (i = 0; i < DATA_SIZE; i++) {
        sum += data[i] * data[i];
    }
    printf("CPU sum is: %d\n", sum);
    printf("Memory bandwidth: %f (MB/s)\n", ((float)(DATA_SIZE * sizeof(int) / 1024 / 1024)) / ((float)max_time_used / (clockRate * 1000)));

    system("pause");

    // return 0;
}