#include "hip/hip_runtime.h"
/*
 * sum_squares_7.cu ������״�ӷ���ʵ�ּӷ����л�
 *
 * @author chenyang li
 */
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <>
#include <hip/device_functions.h>
#include <time.h>

#define DATA_SIZE 1024 * 1024
#define THREAD_NUM 256
#define BLOCK_NUM 32

int data[DATA_SIZE];
int clockRate;

/* ����0-9������� */
void generateNumbers(int *numbers, int size) {
    int i;
    for (i = 0; i < size; i++) {
        numbers[i] = rand() % 10;
    }
}

/* ��ӡGPU�豸��Ϣ */
void printDeviceProps(const hipDeviceProp_t *prop) {
    printf("Device Name: %s\n", prop->name);
    printf("totalGlobalMem: %ld\n", prop->totalGlobalMem);
    printf("sharedMemPerBlock: %d\n", prop->sharedMemPerBlock);
    printf("regsPerBlock: %d\n", prop->regsPerBlock);
    printf("warpSize: %d\n", prop->warpSize);
    printf("memPitch: %d\n", prop->memPitch);
    printf("maxThreadPerBlock: %d\n", prop->maxThreadsPerBlock);
    printf("maxThreadsDim[0-2]: %d %d %d\n", prop->maxThreadsDim[0], prop->maxThreadsDim[1], prop->maxThreadsDim[2]);
    printf("maxGridSize[0-2]: %d %d %d\n", prop->maxGridSize[0], prop->maxGridSize[1], prop->maxGridSize[2]);
    printf("totalConstMem: %d\n", prop->totalConstMem);
    printf("major: %d & minor: %d\n", prop->major, prop->minor);
    printf("clockRate: %d\n", prop->clockRate); clockRate = prop->clockRate;
    printf("textureAlignment: %d\n", prop->textureAlignment);
    printf("deviceOverlap: %d\n", prop->deviceOverlap);
    printf("multiProcessorCount: %d\n", prop->multiProcessorCount);
}

/* CUDA ��ʼ�� */
bool initCUDA() {
    int count, i;
    hipDeviceProp_t prop;

    hipGetDeviceCount(&count);

    if (0 == count) {
        fprintf(stderr, "There is no device.\n");
        return false;
    }

    for (i = 0; i < count; i++) {
        if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
            if (prop.major >= 1) {
                break;
            }
        }
    }

    if (i == count) {
        fprintf(stderr, "There is no device.\n");
        return false;
    }

    hipSetDevice(i);

    printDeviceProps(&prop);

    return true;
}

/* ��������ʱ */
clock_t findMaxTimeUsed(const clock_t *time) {
    int i;
    clock_t min_start = time[0], max_end = time[BLOCK_NUM];
    for (i = 0; i < BLOCK_NUM; i++) {
        if (time[i] < min_start) {
            min_start = time[i];
        }
        if (time[i + BLOCK_NUM] > max_end) {
            max_end = time[i + BLOCK_NUM];
        }
    }

    return max_end - min_start;
}

/* ����ƽ���ͣ�__global__����������GPU��*/
__global__ static void sumOfSquares(int *numbers, int *sub_sum, clock_t *time) {
    int i;

    extern __shared__ int shared[];

    const int block_id = blockIdx.x;
    const int thread_id = threadIdx.x;
    // ���岽���ͼ�������
    int offset, mask;

    if (thread_id == 0) {
        time[block_id] = clock();
    }

    shared[thread_id] = 0;
    for (i = block_id * THREAD_NUM + thread_id; i < DATA_SIZE; i += BLOCK_NUM * THREAD_NUM) {
        shared[thread_id] += numbers[i] * numbers[i];
    }

    if (thread_id == 0) {
        time[block_id + BLOCK_NUM] = clock();
    }

    __syncthreads();

    /* ���мӷ������ */
    offset = 1;
    mask = 1;
    while (offset < THREAD_NUM) {
        // ע�� & �����ȼ�С�� ==
        if ((thread_id & mask) == 0 && thread_id + offset < THREAD_NUM) {
            shared[thread_id] += shared[thread_id + offset];
        }
        offset += offset;
        mask += offset;
        // ÿ����һ����Ҫ�����߳̽���һ��ͬ��
        __syncthreads();
    }

    sub_sum[block_id] = shared[0];
}

int main(void) {
    if (!initCUDA()) {
        return 0;
    }

    int *gpudata;
    int i, sum;
    int sub_sum[BLOCK_NUM], *gpu_sub_sum;
    clock_t time_used[BLOCK_NUM * 2], *gpu_time_used;

    generateNumbers(data, DATA_SIZE);

    hipMalloc((void**)&gpudata, sizeof(int) * DATA_SIZE);
    hipMalloc((void**)&gpu_sub_sum, sizeof(int) * BLOCK_NUM);
    hipMalloc((void**)&gpu_time_used, sizeof(clock_t) * BLOCK_NUM * 2);

    hipMemcpy(gpudata, data, sizeof(int) * DATA_SIZE, hipMemcpyHostToDevice);

    sumOfSquares << < BLOCK_NUM, THREAD_NUM, sizeof(int) * THREAD_NUM >> > (gpudata, gpu_sub_sum, gpu_time_used);

    hipMemcpy(time_used, gpu_time_used, sizeof(clock_t) * BLOCK_NUM * 2, hipMemcpyDeviceToHost);
    hipMemcpy(sub_sum, gpu_sub_sum, sizeof(int) * BLOCK_NUM, hipMemcpyDeviceToHost);

    sum = 0;
    for (i = 0; i < BLOCK_NUM; i++) {
        sum += sub_sum[i];
    }

    hipFree(gpudata);
    hipFree(gpu_sub_sum);
    hipFree(time);

    clock_t max_time_used = findMaxTimeUsed(time_used);
    printf("\nGPU sum is: %d, time used: %f (s)\n", sum, (float)max_time_used / (clockRate * 1000));

    sum = 0;
    for (i = 0; i < DATA_SIZE; i++) {
        sum += data[i] * data[i];
    }
    printf("CPU sum is: %d\n", sum);
    printf("Memory bandwidth: %f (MB/s)\n", ((float)(DATA_SIZE * sizeof(int) / 1024 / 1024)) / ((float)max_time_used / (clockRate * 1000)));

    system("pause");

    // return 0;
}